#include "hip/hip_runtime.h"
#include "mesh_2d.cuh"

#include "common/cuda_memory.cuh"

#include <array>
#include <fstream>
#include <vector>

bool Mesh2D::loadMeshFromFile(const std::string &filename, double scale)
{
    std::ifstream meshFile(filename);

    if(meshFile.is_open()){
        int numVertices, numCells;
        int tmp;
        float tmp2;

        meshFile >> numVertices >> numCells;

        hostVertices.reserve(numVertices);
        hostCells.reserve(numCells);

        for(int i = 0; i < numVertices; ++i){
            Point2 vertex;
            meshFile >> tmp >> vertex.x >> vertex.y >> tmp2;
            hostVertices.push_back({ scale * vertex.x, scale * vertex.y });
        }

        while(!meshFile.eof()){
            meshFile >> tmp >> tmp;
            if(tmp == 203){ //encountered a triangle
                uint3 triangle;
                meshFile >> triangle.x >> triangle.y >> triangle.z;
                
                //indices of vertices are base-1 in the imported files
                triangle.x -= 1;
                triangle.y -= 1;
                triangle.z -= 1;

                hostCells.push_back(triangle);
            } else {        //encountered an entity of another type
                numCells -= 1;
                meshFile >> tmp >> tmp;
            }
        }

        meshFile.close();

        vertices.allocate(numVertices);
        cells.allocate(numCells);

        copy_h2d(hostVertices.data(), vertices.data, vertices.size);
        copy_h2d(hostCells.data(), cells.data, cells.size);

        printf("Loaded mesh with %d vertices and %d cells\n", numVertices, numCells);

        return true;
    } else {
        printf("Error while opening the file\n");
        return false;
    }
}
