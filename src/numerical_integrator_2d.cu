#include "hip/hip_runtime.h"
#include "numerical_integrator_2d.cuh"

#include "common/cuda_helper.cuh"
#include "common/cuda_math.cuh"
#include "common/cuda_memory.cuh"

__device__ Point2 shapeFuncGrad(int i){
    switch (i)
    {
    case 0:
        return { 1.0, 0.0 };
    case 1:
        return { 0.0, 1.0 };
    case 2:
        return { -1.0, -1.0 };
    default:
        return Point2();
    }
}

__device__ double rhsFunction(const Point2& pt) {
    return exp(-(pt.x * pt.x + 10 * pt.y * pt.y));
}

__device__ void addLocalToGlobal(const uint3 &triangle, const double area, const SymmetricMatrix3x3 &localMatrix, double *localRhs,
    const int *rowOffset, const int *colIndices, double *matrixValues, double *rhsVector)
{
    for(int i = 0; i < 3; ++i){
        const unsigned int vertexIndexI = *(&triangle.x + i);

        //elements of the global matrix
        const int indexOfFirstElementInRow = rowOffset[vertexIndexI];

        int numElementsInRow = rowOffset[vertexIndexI + 1] - indexOfFirstElementInRow;
        for(int j = 0; j < 3; ++j){
            const unsigned int vertexIndexJ = *(&triangle.x + j);
            const int index = indexBinarySearch(vertexIndexJ, &colIndices[indexOfFirstElementInRow], numElementsInRow);

            if(index >= 0)
                atomicAdd(&matrixValues[index + indexOfFirstElementInRow], localMatrix(i, j) * area);
        }
        
        //element of the right hand side vector
        atomicAdd(&rhsVector[vertexIndexI], localRhs[i] * area);
    }
}

__global__ void kCalculateCellArea(int n, const Point2 *vertices, const uint3 *cells, double *areas){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        const uint3 triangle = cells[idx];
        const Point2 v12 = vertices[triangle.y] - vertices[triangle.x];
        const Point2 v13 = vertices[triangle.z] - vertices[triangle.x];

        areas[idx] = fabs(cross(v12, v13)) * 0.5;
    }
}

__global__ void kCalculateInvJacobi(int n, const Point2 *vertices, const uint3 *cells, Matrix2x2 *invJacobi){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        const uint3 triangle = cells[idx];
        const Point2 v13 = vertices[triangle.z] - vertices[triangle.x];
        const Point2 v23 = vertices[triangle.z] - vertices[triangle.y];

        Matrix2x2 Jacobi;
        Jacobi(0, 0) = v13.x;   Jacobi(0, 1) = v13.y;
        Jacobi(1, 0) = v23.x;   Jacobi(1, 1) = v23.y;
        
        invJacobi[idx] = Jacobi.inverse();
    }
}

__global__ void kIntegrateOverCell(int n, const Point2 *vertices, const uint3 *cells, double *areas, Matrix2x2 *invJacobi,
    const int *rowOffset, const int *colIndices, double *matrixValues, double *rhsVector,
    const Point3 *qf_coordinates, const double *qf_weights, int qf_points_num)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < n){
        const uint3 triangle = cells[idx];

        const double lambda = 0.5;
        const double area = areas[idx];

        Point2 triangleVertices[3];
        triangleVertices[0] = vertices[triangle.x];
        triangleVertices[1] = vertices[triangle.y];
        triangleVertices[2] = vertices[triangle.z];

        Matrix2x2 cellInvJacobi = invJacobi[idx];

        SymmetricMatrix3x3 localMatrix;
        double localRhs[3] = { 0.0, 0.0, 0.0 };
        double aux;

        for(int k = 0; k < qf_points_num; ++k){
            Point2 quadraturePoint = { 0.0, 0.0 };
            const Point3 Lcoordinates = qf_coordinates[k];
            for (int l = 0; l < 3; ++l)
                quadraturePoint += *(&Lcoordinates.x + l) * triangleVertices[l];

            for(int i = 0; i < 3; ++i){
                for(int j = i; j < 3; ++j){
                    aux = lambda * dot(cellInvJacobi * shapeFuncGrad(i), cellInvJacobi * shapeFuncGrad(j)) * qf_weights[k];

                    localMatrix(i, j) += aux;
                }

                aux = rhsFunction(quadraturePoint) * *(&Lcoordinates.x + i) * qf_weights[k];
                localRhs[i] += aux;
            }
        }

        addLocalToGlobal(triangle, area, localMatrix, localRhs, rowOffset, colIndices, matrixValues, rhsVector);
    }
}

NumericalIntegrator2D::NumericalIntegrator2D(const Mesh2D &mesh_, const QuadratureFormula2D &qf_)
    : mesh(mesh_)
    , qf(qf_)
{
    cellArea.allocate(mesh.getCells().size);
    invJacobi.allocate(mesh.getCells().size);
    unsigned int blocks = blocksForSize(mesh.getCells().size);
    kCalculateCellArea<<<blocks, gpuThreads>>>(mesh.getCells().size, mesh.getVertices().data, mesh.getCells().data, cellArea.data);
    kCalculateInvJacobi<<<blocks, gpuThreads>>>(mesh.getCells().size, mesh.getVertices().data, mesh.getCells().data, invJacobi.data);
}

NumericalIntegrator2D::~NumericalIntegrator2D()
{

}

void NumericalIntegrator2D::assembleSystem(SparseMatrixCSR &csrMatrix, deviceVector<double> &rhsVector)
{
    unsigned int blocks = blocksForSize(mesh.getCells().size);

    kIntegrateOverCell<<<blocks, gpuThreads>>>(mesh.getCells().size, mesh.getVertices().data, mesh.getCells().data, cellArea.data, invJacobi.data,
        csrMatrix.getRowOffset(), csrMatrix.getColIndices(), csrMatrix.getMatrixValues(), rhsVector.data,
        qf.getCoordinates(), qf.getWeights(), qf.getGaussPointsNumber());
}
