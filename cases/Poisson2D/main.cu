#include "hip/hip_runtime.h"
#include "data_export.cuh"
#include "Dirichlet_bcs.cuh"
#include "linear_solver.cuh"
#include "mesh_2d.cuh"
#include "numerical_integrator_2d.cuh"
#include "sparse_matrix.cuh"

#include "common/cuda_math.cuh"
#include "common/gpu_timer.cuh"

#include <vector>

__device__ double rhsFunction(const Point2& pt) {
    return exp(-(pt.x * pt.x + 10 * pt.y * pt.y));
}

__global__ void kIntegrateOverCell(int n, const Point2 *vertices, const uint3 *cells, double *areas, Matrix2x2 *invJacobi,
    const int *rowOffset, const int *colIndices, double *matrixValues, double *rhsVector,
    const Point3 *qf_coordinates, const double *qf_weights, int qf_points_num)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < n){
        const uint3 triangle = cells[idx];

        const double lambda = 0.5;
        const double area = areas[idx];

        Point2 triangleVertices[3];
        triangleVertices[0] = vertices[triangle.x];
        triangleVertices[1] = vertices[triangle.y];
        triangleVertices[2] = vertices[triangle.z];

        Matrix2x2 cellInvJacobi = invJacobi[idx];

        SymmetricMatrix3x3 localMatrix;
        double localRhs[3] = { 0.0, 0.0, 0.0 };
        double aux;

        for(int k = 0; k < qf_points_num; ++k){
            Point2 quadraturePoint = { 0.0, 0.0 };
            const Point3 Lcoordinates = qf_coordinates[k];
            for (int l = 0; l < 3; ++l)
                quadraturePoint += *(&Lcoordinates.x + l) * triangleVertices[l];

            for(int i = 0; i < 3; ++i){
                for(int j = i; j < 3; ++j){
                    aux = lambda * dot(cellInvJacobi * shapeFuncGrad(i), cellInvJacobi * shapeFuncGrad(j)) * qf_weights[k];

                    localMatrix(i, j) += aux;
                }

                aux = rhsFunction(quadraturePoint) * *(&Lcoordinates.x + i) * qf_weights[k];
                localRhs[i] += aux;
            }
        }

        addLocalToGlobal(triangle, area, localMatrix, localRhs, rowOffset, colIndices, matrixValues, rhsVector);
    }
}

class PoissonIntegrator : public NumericalIntegrator2D
{
public:
    PoissonIntegrator(const Mesh2D& mesh_, const QuadratureFormula2D& qf_)
        : NumericalIntegrator2D(mesh_, qf_) { };

	void assembleSystem(SparseMatrixCSR &csrMatrix, deviceVector<double> &rhsVector);
};

void PoissonIntegrator::assembleSystem(SparseMatrixCSR &csrMatrix, deviceVector<double> &rhsVector)
{
    unsigned int blocks = blocksForSize(mesh.getCells().size);

    kIntegrateOverCell<<<blocks, gpuThreads>>>(mesh.getCells().size, mesh.getVertices().data, mesh.getCells().data, cellArea.data, invJacobi.data,
        csrMatrix.getRowOffset(), csrMatrix.getColIndices(), csrMatrix.getMatrixValues(), rhsVector.data,
        qf.getCoordinates(), qf.getWeights(), qf.getGaussPointsNumber());
}

int main(int argc, char *argv[]){
	GpuTimer timer;
    
    timer.start();

    Mesh2D mesh;
    if(!mesh.loadMeshFromFile("../TestProblem2.dat"))
        return EXIT_FAILURE;

    timer.stop("Mesh import");

    const int problemSize = mesh.getVertices().size;

    DirichletBCs bcs;
    
    timer.start();

    {
        std::vector<DirichletNode> hostBcs;

        const auto& vertices = mesh.getHostVertices();

        hostBcs.reserve(0.1 * vertices.size());

        for (unsigned i = 0; i < vertices.size(); ++i) {
            const Point2& node = vertices[i];

            if (std::fabs(node.x - (-1.0)) < CONSTANTS::DOUBLE_MIN)
                hostBcs.push_back({ i, -1.0 });
            else if (std::fabs(node.x - 1.0) < CONSTANTS::DOUBLE_MIN)
                hostBcs.push_back({ i, 1.0 });
            else if (std::fabs(node.y) < CONSTANTS::DOUBLE_MIN)
                hostBcs.push_back({ i, 0.0 });
            else if (std::fabs(node.y - 1.0) < CONSTANTS::DOUBLE_MIN)
                hostBcs.push_back({ i, 2.0 });
        }

        bcs.setupDirichletBCs(hostBcs);
    }

    timer.stop("Boundary conditions setup");

    QuadratureFormula2D qf(1);

    SparseMatrixCSR matrix(mesh);
    PoissonIntegrator integrator(mesh, qf);

    deviceVector<double> rhsVector;
    rhsVector.allocate(problemSize);
    zero_value_device(rhsVector.data, problemSize);

    timer.start();

    integrator.assembleSystem(matrix, rhsVector);
    bcs.applyBCs(matrix, rhsVector);

    timer.stop("Assembly of system and rhs");

    matrix.exportMatrix("matrix.dat");

    deviceVector<double> solution;
    solution.allocate(problemSize);

    timer.start();

    SolverCG cgSolver(1e-8, 1000);
    cgSolver.init(matrix, true);
    cgSolver.solve(matrix, solution, rhsVector);

    timer.stop("PCG solver");
    timer.start();

    SolverGMRES gmresSolver(1e-8, 1000);
    gmresSolver.init(matrix, true);
    gmresSolver.solve(matrix, solution, rhsVector);

    timer.stop("GMRES solver");

    solution.exportToFile("solution.dat");
    rhsVector.exportToFile("rhs.dat");

    DataExport dataExport(mesh);
    dataExport.addScalarDataVector(solution, "solution");
    dataExport.exportToVTK("solution.vtu");

    return EXIT_SUCCESS;
}
